#ifndef __NVCC__
#define __NVCC__
#endif
#include "Network.h"

#include <hiprand.h>
#include <tiny-cuda-nn/config.h>
#include <tiny-cuda-nn/common.h>
#include <tiny-cuda-nn/encoding.h>

using precision_t = tcnn::network_precision_t;
// cuda related
hipStream_t inference_stream;
hipStream_t training_stream;
hiprandGenerator_t rng;

// Position: 3
// Scattered Direction: 2
// Normal: 2
// Roughness: 1
// Diffuse Reflectance: 3
// Specular Reflectance: 3
constexpr static uint32_t NUM_INPUT_DIMS = 14;
// RGB Radiance: 3
constexpr static uint32_t NUM_OUTPUT_DIMS = 3;

struct NeuralRadianceCache::NetworkInternal {
	std::shared_ptr<tcnn::Loss<precision_t>> loss = nullptr;
	std::shared_ptr<tcnn::Optimizer<precision_t>> optimizer = nullptr;
	std::shared_ptr<tcnn::NetworkWithInputEncoding<precision_t>> network = nullptr;
	std::shared_ptr<tcnn::Trainer<float, precision_t, precision_t>> trainer = nullptr;
};

NeuralRadianceCache::NeuralRadianceCache() { m = new NetworkInternal(); }

NeuralRadianceCache::~NeuralRadianceCache() { delete m; }

void NeuralRadianceCache::initialize(PositionEncoding pos_encoding, uint32_t num_hidden_layers, float learning_rate) {
	tcnn::json config = {{"loss", {{"otype", "RelativeL2Luminance"}}},
						 {"optimizer",
						  {{"otype", "EMA"},
						   {"decay", 0.99f},
						   {"nested",
							{
								{"otype", "Adam"},
								{"learning_rate", learning_rate},
								{"beta1", 0.9f},
								{"beta2", 0.99f},
								{"l2_reg", 1e-6f},
							}}}},
						 {"network",
						  {
							  {"otype", "FullyFusedMLP"},
							  {"n_neurons", 64},
							  {"n_hidden_layers", num_hidden_layers},
							  {"activation", "ReLU"},
							  {"output_activation", "None"},
						  }}};
	if (pos_encoding == PositionEncoding::TriangleWave) {
		// config["encoding"] = { {"otype", "NRC"} };
		config["encoding"] = {{"otype", "Composite"},
							  {"nested",
							   {
								   {
									   {"n_dims_to_encode", 3},
									   {"otype", "TriangleWave"},
									   {"n_frequencies", 12},
								   },
								   {
									   {"n_dims_to_encode", 5},
									   {"otype", "OneBlob"},
									   {"n_bins", 4},
								   },
								   {{"n_dims_to_encode", 6}, {"otype", "Identity"}},
							   }}};
		config["optimizer"]["nested"]["epsilon"] = 1e-8f;
	} else if (pos_encoding == PositionEncoding::HashGrid) {
		config["encoding"] = {{"otype", "Composite"},
							  {"nested",
							   {
								   {
									   {"n_dims_to_encode", 3},
									   {"otype", "HashGrid"},
									   {"per_level_scale", 2.0f},
									   {"log2_hashmap_size", 15},
									   {"base_resolution", 16},
									   {"n_levels", 16},
									   {"n_features_per_level", 2},
								   },
								   {
									   {"n_dims_to_encode", 5},
									   {"otype", "OneBlob"},
									   {"n_bins", 4},
								   },
								   {{"n_dims_to_encode", 6}, {"otype", "Identity"}},
							   }}};
		config["optimizer"]["nested"]["epsilon"] = 1e-15f;
	}

	m->loss.reset(tcnn::create_loss<precision_t>(config.value("loss", tcnn::json::object())));
	m->optimizer.reset(tcnn::create_optimizer<precision_t>(config.value("optimizer", tcnn::json::object())));
	m->network = std::make_shared<tcnn::NetworkWithInputEncoding<precision_t>>(
		NUM_INPUT_DIMS, NUM_OUTPUT_DIMS, config.value("encoding", tcnn::json::object()),
		config.value("network", tcnn::json::object()));

	m->trainer = std::make_shared<tcnn::Trainer<float, precision_t, precision_t>>(m->network, m->optimizer, m->loss);

}

void NeuralRadianceCache::finalize() {
	m->trainer = nullptr;
	m->network = nullptr;
	m->optimizer = nullptr;
	m->loss = nullptr;
}

void NeuralRadianceCache::infer(hipStream_t stream, float* input, uint32_t num_data, float* prediction_data) {
	assert((num_data & 0x7F) == 0);
	tcnn::GPUMatrix<float> inputs(input, NUM_INPUT_DIMS, num_data);
	tcnn::GPUMatrix<float> predictions(prediction_data, NUM_OUTPUT_DIMS, num_data);
	m->network->inference(stream, inputs, predictions);
}

void NeuralRadianceCache::train(hipStream_t stream, float* input, float* target_data, uint32_t num_data,
								float* loss_on_cpu) {
	assert((num_data & 0x7F) == 0);
	tcnn::GPUMatrix<float> inputs(input, NUM_INPUT_DIMS, num_data);
	tcnn::GPUMatrix<float> targets(target_data, NUM_OUTPUT_DIMS, num_data);
	auto context = m->trainer->training_step(stream, inputs, targets);
	if (loss_on_cpu) *loss_on_cpu = m->trainer->loss(stream, *context);
}
